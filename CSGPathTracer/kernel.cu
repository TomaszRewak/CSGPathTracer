#include "hip/hip_runtime.h"
#pragma once

#include "kernel.h"

#include "PathTracer/rendering.h"
#include "PathTracer/rendering.h"
#include "PathTracer/scene.h"
#include "Communication/component-mapping.h"

namespace PathTracer
{
	__device__ void copyShapesToSharedMemory(
		Communication::Component* zippedComponents, size_t componentsNumber,
		Component*& components,
		Component**& rootComponents, size_t& rootComponentsNumber)
	{
		if (!threadIdx.x && !threadIdx.y) {
			Communication::mapComponents(
				zippedComponents, componentsNumber,
				components,
				rootComponents, rootComponentsNumber
			);
		}

		__syncthreads();
	}

	__device__ void freeShapes(
		Component* components,
		Component**& rootComponents)
	{
		__syncthreads();

		if (!threadIdx.x && !threadIdx.y) {
			free(components);
			free(rootComponents);
		}
	}

	__global__ void kernel(
		float4* image,
		const size_t imageWidth, const size_t imageHeight,
		size_t scopeX, size_t scopeY, size_t scopes,
		Camera camera,
		Communication::Component* zippedComponents, size_t zippedComponentsNumber,
		size_t frameNumber, unsigned long long seed)
	{
		__shared__ Component* components;
		__shared__ Component** rootComponents; __shared__ size_t rootComponentsNumber;

		copyShapesToSharedMemory(
			zippedComponents, zippedComponentsNumber,
			components,
			rootComponents, rootComponentsNumber);

		PathTracer::Scene scene(rootComponents, rootComponentsNumber);

		size_t x = (blockIdx.x * blockDim.x + threadIdx.x) * scopes + scopeX;
		size_t y = (blockIdx.y * blockDim.y + threadIdx.y) * scopes + scopeY;

		size_t index = y * imageWidth + x;

		hiprandState randState;
		hiprand_init(seed + index * 10, 0, 0, &randState); // worth remembering: hiprand_init(seed + index, 0, 0, &randState);

		Math::Ray ray = camera.getRay(x, y, imageWidth, imageHeight, randState);

		Shading::Color light = Rendering::shootRay<4, 2>(ray, scene, randState);

		if (x < imageWidth && y < imageHeight)
		{
			image[index].x = light.r;
			image[index].y = light.g;
			image[index].z = light.b;
			image[index].w = 1.f / frameNumber;
		}

		freeShapes(components, rootComponents);
	}

	std::default_random_engine generator;

	void renderRect(
		float4* image,
		const size_t imageWidth, const size_t imageHeight,
		Camera camera,
		Communication::Component* zippedComponents, size_t zippedComponentsNumber,
		size_t frameNumber)
	{
		const size_t scopes = 4;

		dim3 block(16, 16, 1);
		dim3 grid(imageWidth / block.x / scopes + 1, imageHeight / block.y / scopes + 1, 1);

		std::uniform_int_distribution<unsigned long long> distribution(0, std::numeric_limits<unsigned long long>::max());
		unsigned long long seed = distribution(generator);

		for (size_t scopeX = 0; scopeX < scopes; scopeX++)
		{
			for (size_t scopeY = 0; scopeY < scopes; scopeY++)
			{
				kernel << <grid, block >> > (
					image,
					imageWidth, imageHeight,
					scopeX, scopeY, scopes,
					camera,
					zippedComponents, zippedComponentsNumber,
					frameNumber, seed);

				hipDeviceSynchronize();
			}
		}
	}
}